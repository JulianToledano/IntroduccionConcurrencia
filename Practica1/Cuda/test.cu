
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512

/***************************************************************
*                        TERMINOLOGÍA                          *
*  Un block puede ser dividido en distintos threads paralelos  *
*  Usamos  threadId.x en vez de blockIdx.x                     *
****************************************************************/

__global__ void add(/*int *a, int *b, int *c*/){
  //c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
  printf("hola desde el kernel");
}

int main(){

  int *a, *b, *c;               // Copias de a b y c en el host
  int *d_a, *d_b, *d_c;         // Copias de a b y c en el device
  int size = N * sizeof(int);

  // Resevamos memoria para las copias en el device
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
//  cudaMalloc((void**)&d_c, size);
/*
  // Obtenemos espacio para las copias de a, b y c dentro del host
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  for(int i = 0; i < N; i++){
    a[i] = i;
    b[i] = i;
  }

  // Copiamos los inputs en el device
  cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);
  cudaMemcpy(d_b, b, size, cudaMemcpyHostToDevice);
*/
  // Lanzamos add kernel con N threads
  add<<<1,N>>>(/*d_a, d_b, d_c*/);

  // Copiamos el resultado a memoria del host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
/*
  for(int i = 0; i < N; i++)
    printf("%d, ", c[i]);*/

  /*// Liberamos memoria
  free(a); free(b); free(c);
  cudaFree(d_a); cudaFree(d_b); cudaFree(d_c);
*/
  return 0;
}
