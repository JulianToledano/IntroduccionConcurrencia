#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "debug_time.h"

#define PROCESADORES 8

typedef struct matriz_t
{
	int filas;
	int columnas;
	int** datos;
} matriz_t;

typedef struct paquete_trabajo{
	int **matriz1_datos;
	int matriz1_inicial;
	int matriz1_final;

	int **matriz2_datos;
	int matriz2_inicial;
	int matriz2_final;
}paquete;


int** crearMatriz(int numFilas, int numColumnas) {
    int** matriz =(int**) malloc(sizeof(int*) * numFilas);
    int i;
    for(i = 0; i < numFilas; i++) {
        matriz[i] = (int*) malloc(sizeof(int) * numColumnas);
    }
    return matriz;
}

int leedato(FILE* fich) {
	//Mejor con memoria dinamica
	char dato[100];
	char datoleido;
	int contador = 0;
	do {
		if(!feof(fich)) {
			fread(&datoleido, sizeof(char), 1, fich);
			if(datoleido != ' ') {
				dato[contador] = datoleido;
				contador++;
			}
		} else datoleido = ' ';
	} while(datoleido != ' ');
	dato[contador] = '\0';
	int datointeger = atoi(dato);
	return datointeger;
}

matriz_t leerMatriz(char* nombreFichero, int traspuesta) {
	matriz_t matriz;
	int numFilas, numColumnas;
	FILE* fich = fopen(nombreFichero, "r");
	if(fich == NULL) {
		printf("Error\n");
		exit(0);
	}
	numFilas = leedato(fich);
	numColumnas = leedato(fich);
	matriz.datos = crearMatriz(numFilas, numColumnas);

	int i, j;
	if(!traspuesta) {
		for(i = 0; i < numFilas; i++) {
	        for(j = 0; j < numColumnas; j++) {
	            matriz.datos[i][j] = leedato(fich);
	        }
    	}
	} else {
		for(i = 0; i < numFilas; i++) {
	        for(j = 0; j < numColumnas; j++) {
	            matriz.datos[j][i] = leedato(fich);
	        }
	    }
	}
	matriz.filas = numFilas;
	matriz.columnas = numColumnas;
	fclose(fich);


    return matriz;
}
/*
int multiplicaVectores(int* v1, int* v2, int size) {
	int result = 0,i;
	for (i = 0; i < size; ++i)
	{
		result += v1[i]*v2[i];
	}
	return result;
}

void multiplicarMatrices(matriz_t m1, matriz_t m2, matriz_t mres) {
	int i, j;
	for(i = 0; i < m1.filas; i++) {
        for(j = 0; j < m2.columnas; j++) {
            mres.datos[i][j] = multiplicaVectores(m1.datos[i], m2.datos[j], m2.columnas);
        }
	}
}
*/
void escribirMatriz(int** matriz, int numFilas, int numColumnas, char* fileName) {
    FILE* fich = fopen(fileName, "w");
    if(fich == NULL) {
        printf("Error\n");
        return;
    }

    char charaux[100];

    sprintf(charaux, "%d %d ", numFilas, numColumnas);
    fwrite(charaux, sizeof(char), strlen(charaux), fich);
    int i,j;
    for(i = 0; i < numFilas; i++) {
        for(j = 0; j < numColumnas; j++) {
            sprintf(charaux, "%d ", matriz[i][j]);
            fwrite(charaux, sizeof(char), strlen(charaux), fich);
        }
    }
    fclose(fich);
}

void printMatrix(matriz_t matrix) {
	int i, j;
	for(i = 0; i < matrix.filas; i++) {
        for(j = 0; j < matrix.columnas; j++) {
            printf("%d ", matrix.datos[i][j]);
        }
        printf("\n");
    }
}

__global__  void multiplicarMatrices(void){
	printf("zdfds");
}

int main(int argc, char** argv) {
/// uso programa: multiplicarMatricesSec <matriz1> <matriz2> <matrizresultado> // Recomendacion una clase matriz

	// cargar datos
	//matriz_t m1, m2, mres;
	int *d_m1;
	/*m1 = leerMatriz(argv[1], 0);
	m2 = leerMatriz(argv[2], 1);*/

	/*int **temp;
	temp = (int**)malloc(sizeof(int*)*m1.filas);*/
	hipMalloc((void**)&d_m1, 10*sizeof(int));
	/*for(int i = 0; i < m1.filas; i++){
		hipMalloc((void**)&(temp[i]),m1.columnas*sizeof(int));
		hipMemcpy(temp[i], m1.datos[i], m1.columnas*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_m1+i, &(temp[i]), sizeof(int*), hipMemcpyHostToDevice);
	}*/

	// reservar resultado
	/*mres.filas = m1.filas;
	mres.columnas = m2.columnas;
	mres.datos = crearMatriz(mres.filas, mres.columnas);*/
	printf("antes");
	multiplicarMatrices<<<2,2>>>();
	printf("dspues");


	// Escribir resultado
	//escribirMatriz(mres.datos, mres.filas, mres.columnas, argv[3]);


	// Imprimir matriz
	//printMatrix(mres);
	// Liberar datos
}
