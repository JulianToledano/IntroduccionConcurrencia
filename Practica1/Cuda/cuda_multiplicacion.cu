#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "debug_time.h"

#define PROCESADORES 8

typedef struct matriz_t
{
	int filas;
	int columnas;
	int** datos;
} matriz_t;

int** crearMatriz(int numFilas, int numColumnas) {
    int** matriz =(int**) malloc(sizeof(int*) * numFilas);
    int i;
    for(i = 0; i < numFilas; i++) {
        matriz[i] = (int*) malloc(sizeof(int) * numColumnas);
    }
    return matriz;
}

int leedato(FILE* fich) {
	//Mejor con memoria dinamica
	char dato[100];
	char datoleido;
	int contador = 0;
	do {
		if(!feof(fich)) {
			fread(&datoleido, sizeof(char), 1, fich);
			if(datoleido != ' ') {
				dato[contador] = datoleido;
				contador++;
			}
		} else datoleido = ' ';
	} while(datoleido != ' ');
	dato[contador] = '\0';
	int datointeger = atoi(dato);
	return datointeger;
}

matriz_t leerMatriz(char* nombreFichero, int traspuesta) {
	matriz_t matriz;
	int numFilas, numColumnas;
	FILE* fich = fopen(nombreFichero, "r");
	if(fich == NULL) {
		printf("Error\n");
		exit(0);
	}
	numFilas = leedato(fich);
	numColumnas = leedato(fich);
	matriz.datos = crearMatriz(numFilas, numColumnas);

	int i, j;
	if(!traspuesta) {
		for(i = 0; i < numFilas; i++) {
	        for(j = 0; j < numColumnas; j++) {
	            matriz.datos[i][j] = leedato(fich);
	        }
    	}
	} else {
		for(i = 0; i < numFilas; i++) {
	        for(j = 0; j < numColumnas; j++) {
	            matriz.datos[j][i] = leedato(fich);
	        }
	    }
	}
	matriz.filas = numFilas;
	matriz.columnas = numColumnas;
	fclose(fich);


    return matriz;
}

void escribirMatriz(int** matriz, int numFilas, int numColumnas, char* fileName) {
    FILE* fich = fopen(fileName, "w");
    if(fich == NULL) {
        printf("Error\n");
        return;
    }

    char charaux[100];

    sprintf(charaux, "%d %d ", numFilas, numColumnas);
    fwrite(charaux, sizeof(char), strlen(charaux), fich);
    int i,j;
    for(i = 0; i < numFilas; i++) {
        for(j = 0; j < numColumnas; j++) {
            sprintf(charaux, "%d ", matriz[i][j]);
            fwrite(charaux, sizeof(char), strlen(charaux), fich);
        }
    }
    fclose(fich);
}

void printMatrix(matriz_t matrix) {
	int i, j;
	for(i = 0; i < matrix.filas; i++) {
        for(j = 0; j < matrix.columnas; j++) {
            printf("%d ", matrix.datos[i][j]);
        }
        printf("\n");
    }
}

__device__ int multiplicarVector(int lado, int* fila, int* columna) {
	int resultado = 0;
	for (int i = 0; i < lado; i++)
		resultado += fila[i] * columna[i];

	return resultado;
}

__global__ void multiplicarMatrices(int tamano, int** matriz1, int** matriz2, int** resultado) {

	int columna = blockIdx.y * blockDim.x + threadIdx.x;
	int fila = blockIdx.x * blockDim.y + threadIdx.y;
	if((columna >= tamano) || (fila>= tamano)) return;
	resultado[fila][columna] = multiplicarVector(tamano, matriz1[fila], matriz2[columna]);
}

int main(int argc, char** argv) {

	DEBUG_TIME_INIT;
	DEBUG_TIME_START;

	// Leer matrices en el host
	matriz_t m1, m2, mres;
	m1 = leerMatriz(argv[1], 0);
	m2 = leerMatriz(argv[2], 1);
	// Reservar memoria para el resultado
	mres.filas = m1.filas;
	mres.columnas = m2.columnas;
	mres.datos = crearMatriz(mres.filas, mres.columnas);

	// Datos de intercambio entre host y device
	int **i_m1, **i_m2, **i_res;
	// Datos del device
	int **d_m1, **d_m2, **d_res;

	i_m1 = (int**)malloc(m1.columnas * sizeof(int*));
	i_m2 = (int**)malloc(m1.columnas * sizeof(int*));
	i_res = (int**)malloc(m1.columnas * sizeof(int*));

	// Reservamos memoria para las matrices en device
	hipMalloc((void**)&d_res,sizeof(int*)* m1.columnas);
	hipMalloc((void**)&d_m1,sizeof(int*)* m1.columnas);
	hipMalloc((void**)&d_m2,sizeof(int*)* m1.columnas);

	// Reservamos memoria para la zona de intercabio
	for(int i = 0; i < m1.columnas; i++){
		hipMalloc((void**)&(i_m1[i]),sizeof(int)* m1.columnas);
		hipMalloc((void**)&(i_m2[i]),sizeof(int)* m1.columnas);
		hipMalloc((void**)&(i_res[i]),sizeof(int)* m1.columnas);
	}
	// Copiamos los datos a la zona de intercambio
	for(int i = 0; i < m1.columnas;i++){
		hipMemcpy(i_m1[i], m1.datos[i], m1.columnas * sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(i_m2[i], m2.datos[i], m1.columnas * sizeof(int),hipMemcpyHostToDevice);
	}
	// Copiamos lso datos de la zona de intercambio a la memoria device
	hipMemcpy(d_m1, i_m1, m1.columnas * sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy(d_m2, i_m2, m1.columnas * sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy(d_res, i_res, m1.columnas * sizeof(int*),hipMemcpyHostToDevice);

	//int bloque = 32;

	dim3 grid = dim3((m1.columnas / 32) + 1, (m1.columnas / 32) + 1, 1);
	dim3 block = dim3(32, 32, 1);
	{
	DEBUG_TIME_INIT;
	DEBUG_TIME_START;
	multiplicarMatrices<<<grid, block>>> (m1.columnas, d_m1, d_m2, d_res);
	DEBUG_TIME_END;
	DEBUG_PRINT_FINALTIME("Tiempo multiplicacion: ");
	}
	for(int i = 0; i < m1.columnas;i++)
		hipMemcpy(mres.datos[i], i_res[i], m1.columnas * sizeof(int), hipMemcpyDeviceToHost);

	// Escribir resultado
	escribirMatriz(mres.datos, mres.filas, mres.columnas, argv[3]);

	//free

	for(int i = 0; i < m1.columnas; i++){
	//	free(matriz1_CPU[i]);
	//	free(matriz2_CPU[i]);
	//	free(resultadoFinal_CPU[i]);

		hipFree(i_m1[i]);
		hipFree(i_m2[i]);
		hipFree(i_res[i]);

	}

	//free(matriz1_CPU);
	//free(matriz2_CPU);
	//free(resultadoFinal_CPU);
	free(i_m1);
	free(i_m2);
	free(i_res);


	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_res);

	DEBUG_TIME_END;
	DEBUG_PRINT_FINALTIME("Tiempo Total: ");

	return 0;
}
