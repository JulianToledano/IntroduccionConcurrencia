
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512

/****************************************************************************
*                             TERMINOLOGÍA                                  *
*  cada invocación paralela de add se llama block                           *
*  el conjunto de blocks se llama grid                                      *
*  cada invocación puede referirse al indice de si bloque con blockIdx.x    *
*****************************************************************************/
__global__ void add(int*a, int*b, int*c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(){
  int *a, *b, *c;           // Copias de a b y c en el host
  int *d_a, *d_b, *d_c;     // Copias de a b y c en el device
  int size = N * sizeof(int);

  // Obtenemos espacio para las copias de a,b y c en device
  hipMalloc((void**) &d_a, size);
  hipMalloc((void**) &d_b, size);
  hipMalloc((void**) &d_c, size);

  // Obtenemos espacio para las copias de a, b y c dentro del host
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  for(int i = 0; i < N; i++){
    a[i] = i;
    b[i] = i;
  }


  c = (int *)malloc(size);

  // Copiamos los imput en device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Lanzamos el kernel add dentro de la GPU
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copiamos los resultados de nuevo en el host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
    printf("%d, ",c[i]);

  // Liberamos memoria
  free(a);free(b);free(c);
  hipFree(d_a);hipFree(d_b);hipFree(d_c);


}
