
#include <hip/hip_runtime.h>
#include <stdio.h>

// add sera ejecuta en el device
// add será llamada desde el host
// add corre en device asi que a,b y c deben apuntar a memoria del device
__global__ void add(int *a, int *b, int *c){
  *c = *a + *b;
}

int main(void){
  int a, b, c;            // Copias de a b y c en el host
  int *d_a, *d_b, *d_c;   // Copias de a, b y c en el device
  int size = sizeof(int);

  // Obtenemos espacio para las copias de a,b y c en device
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Valores input
  a = 2;
  b = 7;

  // Copiamos inputs a device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Lanzamos add() kernen en la GPU
  add<<<1,1>>>(d_a, d_b, d_c);

  // Copiamos el resultado al host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  // Limpiamos
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf("%d",c);
  
  return 0;
}
