
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 510
__global__ void holaMundo() {
  printf("hola desde kernel");
}

int main(void) {

  int *a, *b, *c;               // Copias de a b y c en el host
  int *d_a, *d_b, *d_c;         // Copias de a b y c en el device
  int size = N * sizeof(int);

  // Resevamos memoria para las copias en el device
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, size);

  // Obtenemos espacio para las copias de a, b y c dentro del host
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  for(int i = 0; i < N; i++){
    a[i] = i;
    b[i] = i;
  }

  // Copiamos los inputs en el device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  printf("antes de entrar en gpu\n");
  holaMundo<<<1,N>>>();
  printf("despues de gpu!\n");

  // Copiamos el resultado a memoria del host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
/*
  for(int i = 0; i < N; i++)
    printf("%d, ", c[i]);
*/
  // Liberamos memoria
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
