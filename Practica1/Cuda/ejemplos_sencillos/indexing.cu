// Mezclar threads y bloques

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int*a, int*b, int*c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index < n)
    c[index] = a[index] + b[index];
}

int main(){
  int *a, *b, *c;           // Copias de a b y c en el host
  int *d_a, *d_b, *d_c;     // Copias de a b y c en el device
  int size = N * sizeof(int);

  // Obtenemos espacio para las copias de a,b y c en device
  hipMalloc((void**) &d_a, size);
  hipMalloc((void**) &d_b, size);
  hipMalloc((void**) &d_c, size);

  // Obtenemos espacio para las copias de a, b y c dentro del host
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  for(int i = 0; i < N; i++){
    a[i] = i;
    b[i] = i;
  }
  c = (int *)malloc(size);

  // Copiamos los imput en device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Lanzamos el kernel add dentro de la GPU
  add<<<(N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c,N);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
    printf("%d, ",c[i]);

  // Liberamos memoria
  free(a);free(b);free(c);
  hipFree(d_a);hipFree(d_b);hipFree(d_c);


}
